#include "hip/hip_runtime.h"
// MP 4 Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];
// Due Tuesday, January 15, 2013 at 11:59 p.m. PST
#include    <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

__global__ void total(float * input, float * output, int len) {
    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index
}

int main(int argc, char ** argv) {
    int ii;
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numInputElements; // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numInputElements);

    numOutputElements = numInputElements / (BLOCK_SIZE<<1);
    if (numInputElements % (BLOCK_SIZE<<1)) {
        numOutputElements++;
    }
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));

    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    //@@ Initialize the grid and block dimensions here

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here

    wbTime_stop(Copy, "Copying output memory to the CPU");

    /********************************************************************
     * Reduce output vector on the host
     * NOTE: One could also perform the reduction of the output vector
     * recursively and support any size input. For simplicity, we do not
     * require that for this lab.
     ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] += hostOutput[ii];
    }

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, 1);

    free(hostInput);
    free(hostOutput);

    return 0;
}
