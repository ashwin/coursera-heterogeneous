#include "hip/hip_runtime.h"
// MP 5 Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}
// Due Tuesday, January 22, 2013 at 11:59 p.m. PST

#include    <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

__global__ void scan(float * input, float * output, int len) {
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*) malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements*sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements*sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceInput, 0, numElements*sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements*sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements*sizeof(float), hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}


